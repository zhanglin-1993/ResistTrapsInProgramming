
#include <hip/hip_runtime.h>
//
// atomic operation
//
/*
/
/1.when the variable is process by the multi-threads, conllision could be happened when the specific variable is read and wrote by multi-threads,
/at the time, the conllision can be resolved by atomic operation, in which the variable is locked when it was read and wrote by one thread.
/
/2.for cuda 11.1, the atomic operation has been provided for long long time. when cuda runtime and toolkit is installed correctly, then the 
/atomic operation, e.g. atomicAdd is already provided, the function can be used directly. what should pay attention to include:
/
/2.1 the project is built on cuda, otherwise  2.2 and 2.3 is not seen:
//TODO    right click project - build dependencies - build customizations - cuda 11.1
/2.2 the cuda file using the atomic operation should be include for build and the item type should be cuda file:
//TODO    right click the cuda file - properties - item type - cuda c/c++
/2.3 different operation capbilities are supported by different gpu, i.e., atomicAdd is only supported by gpu with capabilities no smaller than 6.0
/and thus, for the specific atomoic operation, the operation capabilities should be configured:
//TODO   right click project - properties - cuda c/c++ - device - compute_60,sm_60
/2.4 another point should be clarified is that, the atomic operation might be optimized by the NVIDIA,
/but the precondition is the vector which would be operated has been set values before. if the atomic operation is applied in a function in which 
/the vector value is calculated one by one first, then the atomic operation could be inefficient.
/
*/