
#include <hip/hip_runtime.h>
//
// char* and float* variable
//
/*
/
/1.when data was processed by functions in C++, the pointer could be passed to the function as a paramter.
/And thus, only a constant variable indicating the success or failure of the function is needed to called back,
/then, parameters or data processed are not needed to passed to the calling function.
/
/2.for example, when a float variable is needed to be processed, then a pointer float* can be passed to the called function,
/assumpt another function which process a double variable with the same processing code is needed,
/similar function can be wrote with double* pointer, while longer codes should be added.
/
/3.better method can be applied when char* pointer but not the float* or double* pointer is passed to the function,
/and meanwhile, ransfer the char* pointer to the float* or double* pointer in the function, then the requirement for
/these two kind of variable could be satisfied.
/
/4.the reason lies in the reality that the float or double variable is stored binarily in the memory, the difference
/are only the length read once for the variable. for example, if a char* variable is passed to the function, it should
/be transfered to the float* variable, while if not, char variable with length 1/4 of that in float is read,
/and also the variable read is comletely different from the float one.
/
*/